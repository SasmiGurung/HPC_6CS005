#include "hip/hip_runtime.h"
#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>

/********************************************************************************
  This CUDA program demonstrates how do process image using CUDA. This program
  will take image and performs Gaussian Blur filtering then saves the blurred 
  image. 

  Compile with:

    nvcc task3_partB.cu -o task3_partB lodepng.cpp

  To run:
  
    ./task3_partB "input filename"

  Author: Sasmita Gurung 
  University Email: S.Gurung12@wlv.ac.uk
**********************************************************************************/

__device__ unsigned int deviceWidth;

//Getting Red pixels
__device__ unsigned char getRed(unsigned char *image, unsigned int row, unsigned int column)
{
    unsigned int i = (row * deviceWidth * 4) + (column * 4);

    return image[i];
}

//Getting Green pixels
__device__ unsigned char getGreen(unsigned char *image, unsigned int row, unsigned int column)
{
    unsigned int i = (row * deviceWidth * 4) + (column * 4) + 1;

    return image[i];
}

//Getting Blue  pixels
__device__ unsigned char getBlue(unsigned char *image, unsigned int row, unsigned int column)
{
    unsigned int i = (row * deviceWidth * 4) + (column * 4) + 2;

    return image[i];
}

//Getting Alpha pixels (transperancy)
__device__ unsigned char getAlpha(unsigned char *image, unsigned int row, unsigned int column)
{
    unsigned int i = (row * deviceWidth * 4) + (column * 4) + 3;

    return image[i];
}

//Setting Red value
__device__ void setRed(unsigned char *image, unsigned int row, unsigned int column, unsigned char red)
{
    unsigned int i = (row * deviceWidth * 4) + (column * 4);

    image[i] = red;
}

//Setting Green value
__device__ void setGreen(unsigned char *image, unsigned int row, unsigned int column, unsigned char green)
{
    unsigned int i = (row * deviceWidth * 4) + (column * 4) + 1;

    image[i] = green;
}

//Setting Blue value
__device__ void setBlue(unsigned char *image, unsigned int row, unsigned int column, unsigned char blue)
{
    unsigned int i = (row * deviceWidth * 4) + (column * 4) + 2;

    image[i] = blue;
}

//Setting Alpha value
__device__ void setAlpha(unsigned char *image, unsigned int row, unsigned int column, unsigned char alpha)
{
    unsigned int i = (row * deviceWidth * 4) + (column * 4) + 3;

    image[i] = alpha;
}


__global__ void applyGaussianBlurr(unsigned char* image, unsigned char* newImage, unsigned int *width){
    int row = blockIdx.x+1;
    int column = threadIdx.x+1;

    deviceWidth = *width;
    
    unsigned redTL, redTC, redTR;
    unsigned redL, redC, redR;
    unsigned redBL, redBC, redBR;
    unsigned newRed;

    unsigned greenTL, greenTC, greenTR;
    unsigned greenL, greenC, greenR;
    unsigned greenBL, greenBC, greenBR;
    unsigned newGreen;

    unsigned blueTL, blueTC, blueTR;
    unsigned blueL, blueC, blueR;
    unsigned blueBL, blueBC, blueBR;
    unsigned newBlue;
    
    setGreen(newImage, row, column, getGreen(image, row, column));
    setBlue(newImage, row, column, getBlue(image, row, column));
    setAlpha(newImage, row, column, 255);

    redTL = getRed(image, row - 1, column - 1);
    redTC = getRed(image, row - 1, column);
    redTR = getRed(image, row - 1, column + 1);

    redL = getRed(image, row, column - 1);
    redC = getRed(image, row, column);
    redR = getRed(image, row, column + 1);

    redBL = getRed(image, row + 1, column - 1);
    redBC = getRed(image, row + 1, column);
    redBR = getRed(image, row + 1, column + 1);

    newRed = (redTL+redTC+redTR+redL+redC+redR+redBL+redBC+redBR)/9;  //Bluring red columnor value

    setRed(newImage, row, column, newRed);

    greenTL = getGreen(image, row - 1, column - 1);
    greenTC = getGreen(image, row - 1, column);
    greenTR = getGreen(image, row - 1, column + 1);

    greenL = getGreen(image, row, column - 1);
    greenC = getGreen(image, row, column);
    greenR = getGreen(image, row, column + 1);

    greenBL = getGreen(image, row + 1, column - 1);
    greenBC = getGreen(image, row + 1, column);
    greenBR = getGreen(image, row + 1, column + 1);

    newGreen = (greenTL+greenTC+greenTR+greenL+greenC+greenR+greenBL+greenBC+greenBR)/9; //Bluring green columnor value

    setGreen(newImage, row, column, newGreen);

    blueTL = getBlue(image, row - 1, column - 1);
    blueTC = getBlue(image, row - 1, column);
    blueTR = getBlue(image, row - 1, column + 1);

    blueL = getBlue(image, row, column - 1);
    blueC = getBlue(image, row, column);
    blueR = getBlue(image, row, column + 1);

    blueBL = getBlue(image, row + 1, column - 1);
    blueBC = getBlue(image, row + 1, column);
    blueBR = getBlue(image, row + 1, column + 1);

    newBlue = (blueTL+blueTC+blueTR+blueL+blueC+blueR+blueBL+blueBC+blueBR)/9; //Bluring blue columnor value

    setBlue(newImage, row, column, newBlue);

}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference) {
    long long int ds =  finish->tv_sec - start->tv_sec; 
    long long int dn =  finish->tv_nsec - start->tv_nsec; 

    if(dn < 0 ) {
        ds--;
        dn += 1000000000; 
    } 
    *difference = ds * 1000000000 + dn;

    return !(*difference > 0);
}

int main(int argc, char **argv)
{
    unsigned char *image;
    const char *filename = argv[1];
    const char *newFileName = "blurred_image.png";
    unsigned char *newImage;
    unsigned int height = 0, width = 0;
        
    //Decoding Image	
    lodepng_decode32_file(&image, &width, &height, filename);
    newImage = (unsigned char *)malloc(height * width * 4 * sizeof(unsigned char));

    //Declaring gpuImage and setting the value
    unsigned char * gpuImage;
    hipMalloc( (void**) &gpuImage, sizeof(char) * height*width*4); 
    hipMemcpy(gpuImage, image, sizeof(char) *  height*width*4, hipMemcpyHostToDevice);

    //Declaring gpuNewImage 
    unsigned char * gpuNewImage;
    hipMalloc( (void**) &gpuNewImage, sizeof(char) * height*width*4);

    //Declaring gpuImageWidth and setting the value 
    unsigned int* gpuWidth; 
    hipMalloc( (void**) &gpuWidth, sizeof(int));
    hipMemcpy(gpuWidth, &width, sizeof(int), hipMemcpyHostToDevice);

    struct timespec start, finish;
    long long int time_elapsed;
    clock_gettime(CLOCK_MONOTONIC, &start);

    applyGaussianBlurr<<<height-1,width-1>>>(gpuImage, gpuNewImage, gpuWidth);
    hipDeviceSynchronize();

    printf("Image width = %d, height = %d\n", width, height);

    clock_gettime(CLOCK_MONOTONIC, &finish);
    time_difference(&start, &finish, &time_elapsed);
    printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

    //Getting newImage data from gpu
    hipMemcpy(newImage, gpuNewImage, sizeof(char) * height * width * 4, hipMemcpyDeviceToHost);

    //Encoding image 
    lodepng_encode32_file(newFileName, newImage, width, height);
    return 0;
}
