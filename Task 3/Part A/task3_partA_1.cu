
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/********************************************************************************
  This CUDA program demonstrates how to crack an encrypted password using a simple
  "brute force" algorithm. In this program. In this program a password consisting
  of two uppercase letters and two digit integers are cracked.

  Compile with:

    nvcc task3_partA_1.cu -o task3_partA_1

  To run:
  
    ./task3_partA_1 "password"

  If you want to analyse the results then use the redirection operator to send
  output to a file that you can be viewed using an editor

    ./task3_partA_1 "password" > task3_partA_1_results.txt

  Author: Sasmita Gurung 
  University Email: S.Gurung12@wlv.ac.uk
**********************************************************************************/

__device__ char* EncryptPassword(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;  
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;  
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
        //checking all upper case letter limits
        if(i >= 0 && i < 6){
			if(newPassword[i] > 90){
				newPassword[i] = (newPassword[i] - 90) + 65;
			}else if(newPassword[i] < 65){
				newPassword[i] = (65 - newPassword[i]) + 65;
			}
		}
        //checking number section
        else{
            if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}

    // Returns encyted password
	return newPassword; 
}

__device__ int compareStrings(char* stringOne, char* stringTwo){
	
    while(*stringOne)
    {
        //Comparing the two strings
        if (*stringOne != *stringTwo)
            break;
 
        //Changing Pointer location
        stringOne++;
        stringTwo++;
    }
 
    // Returing the 0 if the two strings matches 
    return *(const unsigned char*)stringOne - *(const unsigned char*)stringTwo;
}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference) {
    long long int ds =  finish->tv_sec - start->tv_sec; 
    long long int dn =  finish->tv_nsec - start->tv_nsec; 

    if(dn < 0 ) {
        ds--;
        dn += 1000000000; 
    } 
    *difference = ds * 1000000000 + dn;
    return !(*difference > 0);
}

__global__ void kernel(char * alphabet, char * numbers, char * rawPassword){

    char generatedRawPassword[4];

    generatedRawPassword[0] = alphabet[blockIdx.x];
    generatedRawPassword[1] = alphabet[blockIdx.y];

    generatedRawPassword[2] = numbers[threadIdx.x];
    generatedRawPassword[3] = numbers[threadIdx.y];

    //Raw Password being encrypted
    char *encPassword = EncryptPassword(rawPassword);
        
    //Comparing encrypted generated password with encrypted password 
    if(compareStrings(EncryptPassword(generatedRawPassword), encPassword) == 0){
        printf("Your password is cracked : %s = %s\n", generatedRawPassword, rawPassword);
        printf("Your password Encrypted password : %s = %s\n", encPassword);
        }
}

int main(int argc, char ** argv){

    char cpuAlphabet[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
    char cpuNumbers[10] = {'0','1','2','3','4','5','6','7','8','9'};

    char * gpuAlphabet;
    hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
    hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

    char * gpuNumbers;
    hipMalloc( (void**) &gpuNumbers, sizeof(char) * 10); 
    hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);

    char * password;
    hipMalloc( (void**) &password, sizeof(char) * 26); 
    hipMemcpy(password, argv[1], sizeof(char) * 26, hipMemcpyHostToDevice);

    struct timespec start, finish;
    long long int time_elapsed;
    clock_gettime(CLOCK_MONOTONIC, &start);
        
    kernel<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers, password);
    hipDeviceSynchronize();

    clock_gettime(CLOCK_MONOTONIC, &finish);
    time_difference(&start, &finish, &time_elapsed);
    printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

    return 0;
}