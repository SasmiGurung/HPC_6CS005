
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/********************************************************************************
  This CUDA program demonstrates how to crack an encrypted password using a simple
  "brute force" algorithm. In this program. In this program a password consisting
  of two uppercase letters and two digit integers are cracked.

  Compile with:

    nvcc task3_partA_1.cu -o task3_partA_1

  To run:
  
    ./task3_partA_1

  If you want to analyse the results then use the redirection operator to send
  output to a file that you can be viewed using an editor

    ./task3_partA_1 > task3_partA_1_results.txt

  Author: Sasmita Gurung 
  University Email: S.Gurung12@wlv.ac.uk
**********************************************************************************/

__device__ int match(char *check) {
  char plainPassword_1[] = "BF9999";
  char plainPassword_2[] = "CN9898";
  char plainPassword_3[] = "BT9893";
  char plainPassword_4[] = "MA5369";

  char *a = check;
  char *b = check;
  char *c = check;
  char *d = check;
  char *p1 = plainPassword_1;
  char *p2 = plainPassword_2;
  char *p3 = plainPassword_3;
  char *p4 = plainPassword_4;

  while(*a == *p1) { 
   if(*a == '\0') 
    {
      printf("(Found) Password cracked is: %s\n",plainPassword_1);
      break;
    }

    a++;
    p1++;
  }
	
  while(*b == *p2) { 
   if(*b == '\0') 
    {
      printf("(Found) Password cracked is: %s\n",plainPassword_2);
      break;
    }

    b++;
    p2++;
  }

  while(*c == *p3) { 
   if(*c == '\0') 
    {
      printf("(Found) Password cracked is: %s\n",plainPassword_3);
      break;
    }

    c++;
    p3++;
  }

  while(*d == *p4) { 
   if(*d == '\0') 
    {
      printf("(Found) Password cracked is: %s\n",plainPassword_4);
      return 1;
    }

    d++;
    p4++;
  }
  return 0;

}


__global__ void  kernel() {
  char w,x,y,z;
  char password[7];
  password[6] = '\0';

  int i = blockIdx.x+65;
  int j = threadIdx.x+65;
  char firstValue = i; 
  char secondValue = j; 
    
password[0] = firstValue;
password[1] = secondValue;

for(w='0'; w<='9'; w++){
  for(x='0'; x<='9'; x++){
    for(y='0'; y<='9'; y++){
      for(z='0'; z<='9'; z++){
	  password[2] = w;
	  password[3] = x;
	  password[4] = y;
	  password[5] = z; 
	  if(match(password)) {
	    printf("password found: %s\n", password);  
	  } 
          else {
	     	  printf("(Processing) Brute Force Tried: %s\n", password); 
	      }
	   }
	}
    }
  }

}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}


